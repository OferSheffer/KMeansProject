#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <omp.h>
#include <math.h>
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include "Kmeans.h"


#define CHKMAL_ERROR	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
#define CHKMEMCPY_ERROR if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
#define CHKSYNC_ERROR	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize failed! Error code %d\n", cudaStatus); goto Error; }
#define EVENT_ERROR		if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaEventOperation failed! Error code %d\n", cudaStatus); goto Error; }

// arrSize indices; THREADS_PER_BLOCK * NO_BLOCKS total threads;
// Each thread in charge of THREAD_BLOCK_SIZE contigeous indices

#define MASTER 0
#define THREADS_PER_BLOCK 1024
#define NEW_JOB 0
#define STOP_WORKING 1

__global__ void reClusterWithCuda(xyArrays* d_kCenters, const int ksize, xyArrays* d_xya, int* pka, bool* d_kaFlags, const int size)
{
	__shared__ bool dShared_kaFlags[THREADS_PER_BLOCK]; // array to flag changes in point-to-cluster association

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int prevPka;
	// for every point: save idx where min(distance from k[idx]	
	if (tid < size)
	{
		dShared_kaFlags[tid] = false; // no changes yet
		prevPka = pka[tid]; // save associated cluster idx
		float minSquareDist = INFINITY;
		float curSquareDist;
		for (int idx = 0; idx < ksize; idx++)
		{
			curSquareDist = powf(d_xya->x[tid] - d_kCenters->x[idx], 2) + powf(d_xya->y[tid] - d_kCenters->y[idx], 2);
			if (curSquareDist < minSquareDist)
			{
				minSquareDist = curSquareDist;
				pka[tid] = idx;
			}
		}
		if (pka[tid] != prevPka)
		{
			dShared_kaFlags[tid] = true;
		}
		// reduction for d_kaFlag
		__syncthreads();
		// do reduction in shared mem
		//reduce(dShared_kaFlags);
		// each thread loads one element from global to shared mem
		unsigned int ltid = threadIdx.x;
#if 0
		unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
		blockIdx.x * blockDim.x + threadIdx.x;

		if (i < size) dShared_kaFlags[ltid] = dShared_kaFlags[i] | dShared_kaFlags[i + blockDim.x];
		__syncthreads();
#endif
		// do reduction in shared mem
		for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
		{
			if (tid < s)
				dShared_kaFlags[ltid] |= dShared_kaFlags[ltid + s];
			__syncthreads();
		}
		if (ltid < 32) //unroll warp
		{
			dShared_kaFlags[ltid] += dShared_kaFlags[ltid + 32];
			dShared_kaFlags[ltid] += dShared_kaFlags[ltid + 16];
			dShared_kaFlags[ltid] += dShared_kaFlags[ltid + 8];
			dShared_kaFlags[ltid] += dShared_kaFlags[ltid + 4];
			dShared_kaFlags[ltid] += dShared_kaFlags[ltid + 2];
			dShared_kaFlags[ltid] += dShared_kaFlags[ltid + 1];
		}

		// write result for this block to global mem
		if (tid == 0) d_kaFlags[blockIdx.x] = dShared_kaFlags[0];
	}
}

// Helper for kDiamBlockWithCuda
__device__ void AtomicMax(float * const address, const float value)
{
	if (*address >= value)
	{
		return;
	}

	int * const address_as_i = (int *)address;
	int old = *address_as_i, assumed;

	do
	{
		assumed = old;
		if (__int_as_float(assumed) >= value)
		{
			break;
		}

		old = atomicCAS(address_as_i, assumed, __float_as_int(value));
	} while (assumed != old);
}

//Note: will be x2 faster with smaller blocks -- but will require (^2/numproc) runs
__global__ void kDiamBlockWithCuda(float* kDiameters, const int ksize, xyArrays* d_xya, int* pka, const int size, const int blkAIdx, const int blkBIdx)
{
	__shared__ float dShared_SquaredXYAB[THREADS_PER_BLOCK * 4]; // save squared values for reuse

	// local shared mem speedup - save squared values for reuse
	// diameter^2 = (XA-XB)^2 + (YA-YB)^2 = XA^2+XB^2+YA^2+YB^2  -2*XA*XB -2*YA*YB
	unsigned int tidA = blkAIdx * blockDim.x + threadIdx.x;
	unsigned int tidB = blkBIdx * blockDim.x + threadIdx.x;
	dShared_SquaredXYAB[4 * threadIdx.x] = powf(d_xya->x[tidA], 2);	// i%4==0: x^2 of blkA
	dShared_SquaredXYAB[4 * threadIdx.x + 1] = powf(d_xya->x[tidB], 2);	// i%4==1: x^2 of blkB
	dShared_SquaredXYAB[4 * threadIdx.x + 2] = powf(d_xya->y[tidA], 2);	// i%4==2: y^2 of blkA
	dShared_SquaredXYAB[4 * threadIdx.x + 3] = powf(d_xya->y[tidB], 2);	// i%4==3: y^2 of blkB
	__syncthreads();

	float max = 0;
	float cur;

	// run kernel with a single block, use external block indices to syncronize operations
	for (int oIdx = 0; oIdx < blockDim.x; oIdx++)
	{
		// prevent repeated calculations
		if (threadIdx.x < oIdx)
		{
			tidB = blkBIdx * blockDim.x + oIdx;

			// only calculate for points with the same k association
			if (pka[tidA] == pka[tidB])
			{
				// XA^2+XB^2+YA^2+YB^2  -2*XA*XB -2*YA*YB
				cur = dShared_SquaredXYAB[4 * threadIdx.x]     + dShared_SquaredXYAB[4 * oIdx + 1]
					+ dShared_SquaredXYAB[4 * threadIdx.x + 2] + dShared_SquaredXYAB[4 * oIdx + 3]
					- 2 * d_xya->x[tidA] * d_xya->x[tidB] - 2 * d_xya->y[tidA] * d_xya->y[tidA];
				if (cur > max) max = cur;
			}
		}
	}
	//TODO: consider reduction instead
	// takes advantage of varying completion times of threads
	AtomicMax(&(kDiameters[pka[threadIdx.x]]), max);
}





// Helper function for finding best centers for ksize clusters
hipError_t kCentersWithCuda(xyArrays* kCenters, int ksize, xyArrays* xya, int* pka, long N, int LIMIT)
{
	hipError_t cudaStatus;
	const int NO_BLOCKS = (N % THREADS_PER_BLOCK == 0) ? N / THREADS_PER_BLOCK : N / THREADS_PER_BLOCK + 1;
	const int THREAD_BLOCK_SIZE = THREADS_PER_BLOCK;

	// data size protection code
	/*
	if (N % (THREADS_PER_BLOCK * NO_BLOCKS) != 0) {
	fprintf(stderr, "reClusterWithCuda launch failed:\n"
	"Array size (%d) modulo Total threads (%d) != 0.\n"
	"Try changing number of threads.\n", N, (THREADS_PER_BLOCK * NO_BLOCKS));
	goto Error;
	} */
	initK(ksize);				// K-centers = first points in data (on host)
	int iter = 0;
	size_t SharedMemBytes = N * sizeof(bool); // shared memory for flag work
	bool flag;

	// memory init block
	//{
	size_t nDataBytes = N * sizeof(*xya);  // N x 2 x sizeof(float)
	size_t nKCenterBytes = ksize * sizeof(*kCenters);
	bool	 *h_kaFlags;
	int	 *d_pka;					// array to associate xya points with their closest cluster
	bool     *d_kaFlags;				// array to flags changes in point-to-cluster association	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// allocate host-side helpers
	h_kaFlags = (bool*)malloc(NO_BLOCKS * sizeof(bool));

	// allocate device memory
	xyArrays *d_xya,
		*d_kCenters;				// data and k-centers xy information
	xyArrays da_xya, h_kCenters;     // da_xya device anchor for copying xy-arrays data

	hipMalloc(&d_xya, sizeof(xyArrays)); CHKMAL_ERROR;

	hipMalloc(&(da_xya.x), nDataBytes / 2); CHKMAL_ERROR;
	hipMalloc(&(da_xya.y), nDataBytes / 2); CHKMAL_ERROR;
	hipMemcpy(da_xya.x, xya->x, nDataBytes / 2, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
	hipMemcpy(da_xya.y, xya->y, nDataBytes / 2, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;

	hipMalloc(&d_kCenters, sizeof(xyArrays));
	hipMalloc(&(h_kCenters.x), nKCenterBytes / 2); CHKMAL_ERROR;
	hipMalloc(&(h_kCenters.y), nKCenterBytes / 2); CHKMAL_ERROR;
	
	hipMemcpy(d_xya, &da_xya, sizeof(xyArrays), hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
	hipMemcpy(d_kCenters, &h_kCenters, sizeof(xyArrays), hipMemcpyHostToDevice); CHKMEMCPY_ERROR;

	hipMalloc(&d_pka, N * sizeof(int)); CHKMAL_ERROR;
	hipMalloc(&d_kaFlags, N * sizeof(bool)); CHKMAL_ERROR;

	// copy cluster association data from host to device
	hipMemcpy(d_pka, pka, N * sizeof(int), hipMemcpyHostToDevice); CHKMEMCPY_ERROR;

	cudaStatus = hipMemset((void*)d_kaFlags, 0, N * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!\n");
		goto Error;
	}
	//}

	// *** phase 1 ***
	do {
		
		hipMemcpy(h_kCenters.x, kCenters->x, nKCenterBytes / 2, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
		hipMemcpy(h_kCenters.y, kCenters->y, nKCenterBytes / 2, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
		//TEST KCenters per iteration
		/*
		for (int i = 0; i < ksize; i++)
		{
			printf("%d: k%d, %8.3f, %8.3f\n", iter+1, i, kCenters->x[i], kCenters->y[i]);
		}
		*/

		//KernelFunc << <DimGrid, DimBlock, SharedMemBytes >> >
		reClusterWithCuda << <NO_BLOCKS, THREADS_PER_BLOCK, SharedMemBytes >> > (d_kCenters, ksize, d_xya, d_pka, d_kaFlags, N); // THREADS_PER_BLOCK, THREAD_BLOCK_SIZE
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reClusterWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipDeviceSynchronize(); CHKSYNC_ERROR;


		cudaStatus = hipMemcpy(h_kaFlags, d_kaFlags, NO_BLOCKS * sizeof(bool), hipMemcpyDeviceToHost); CHKMEMCPY_ERROR;
		cudaStatus = hipMemcpy(pka, d_pka, N * sizeof(int), hipMemcpyDeviceToHost); CHKMEMCPY_ERROR;
		
		flag = ompReduceCudaFlags(h_kaFlags, NO_BLOCKS);
		
		//TODO: consider replacing with a CUDA implementation
		ompRecenterFromCuda(ksize);

	} while (++iter < LIMIT && flag);  // association changes: need to re-cluster

	//TODO: use if using CUDA to reCenter
	//hipMemcpy(kCenters->x, h_kCenters.x, nKCenterBytes / 2, hipMemcpyDeviceToHost); CHKMEMCPY_ERROR;
	//hipMemcpy(kCenters->y, h_kCenters.y, nKCenterBytes / 2, hipMemcpyDeviceToHost); CHKMEMCPY_ERROR;

	//TODO quick test
	printf("k-complete:\n");
	for (int i = 0; i < ksize; i++)
	{
		printf("%d, %8.3f, %8.3f\n", i, kCenters->x[i], kCenters->y[i]);
	}
	

	free(h_kaFlags);

Error:
	hipFree(d_xya);
	hipFree(d_kCenters);
	hipFree(d_pka);
	hipFree(d_kaFlags);

	return cudaStatus;
}

// Helper function for obtaining best candidates for kDiameters on a block x block metric
hipError_t kDiametersWithCuda(float* kDiameters, int ksize, xyArrays* xya, int* pka, long N, int myid, int numprocs)
{
	hipError_t cudaStatus = hipSuccess; //TODO: rm success
	const int NO_BLOCKS = (N % THREADS_PER_BLOCK == 0) ? N / THREADS_PER_BLOCK : N / THREADS_PER_BLOCK + 1;
	const int THREAD_BLOCK_SIZE = THREADS_PER_BLOCK;
	MPI_Status status;

	for (int i = 0; i < ksize; i++)
	{
		kDiameters[i] = 0;
	}

	// allocate device memory
	size_t nDataBytes = N * sizeof(*xya);  // N x 2 x sizeof(float)
	xyArrays da_xya;  // device anchor for copying xy-arrays data
	hipMalloc(&(da_xya.x), nDataBytes / 2); CHKMAL_ERROR;
	hipMalloc(&(da_xya.y), nDataBytes / 2); CHKMAL_ERROR;
	hipMemcpy(da_xya.x, xya->x, nDataBytes / 2, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
	hipMemcpy(da_xya.y, xya->y, nDataBytes / 2, hipMemcpyHostToDevice); CHKMEMCPY_ERROR; 
	xyArrays *d_xya;
	hipMalloc(&d_xya, sizeof(xyArrays));
	hipMemcpy(d_xya, &da_xya, sizeof(xyArrays), hipMemcpyHostToDevice); CHKMEMCPY_ERROR;




	//MPI test
	if (myid == MASTER)
		printf("id %d, %3d: %f, %d\n", myid, NO_BLOCKS, kDiameters[0], ksize); fflush(stdout);


	//MPI single -- working out the single BLOCK problem with CUDA
	if (myid == MASTER)
	{
		//TEST kDiameters 
		for (int i = 0; i < ksize; i++)
		{
			printf("kDiam%d: %8.3f\n", i, kDiameters[i]); fflush(stdout);
		}
		

		kDiamBlockWithCuda(d_kDiameters, ksize, d_xya, d_pka, size, 0, 0);

		//TEST kDiameters 
		for (int i = 0; i < ksize; i++)
		{
			printf("kDiam%d: %8.3f\n", i, kDiameters[i]); fflush(stdout);
		}


	}

	//MASTER-SLAVES
	/*
	if (myid == MASTER)
	{
		int fact = 1, x;
		for (int c = 2; c <= NO_BLOCKS; c++)
			fact *= c;
		int* jobs = initJobArray(NO_BLOCKS, fact);
		int resultsCounter = 0;
		
		//async initializations for MASTER
		hipEvent_t myJobIsDone;
		cudaStatus = hipEventCreateWithFlags(&myJobIsDone, hipEventDisableTiming); EVENT_ERROR;
		hipEventDestroy(myJobIsDone); EVENT_ERROR;

		//use MASTER GPU to asynchronously run first job
		//TODO:
		//hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
		//increment_kernel << <blocks, threads, 0, 0 >> >(d_a, value);
		//hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
		//hipEventRecord(myJobIsDone, 0);
		//
		//while (hipEventQuery(stop) == hipErrorNotReady) {
			//TODO:
			//non-blocking recv from slaves;
		// }
		
		// distribute work to SLAVES
		for (x = 1; x < numprocs && x < fact; x++)
		{
			// send numprocs values to get the work started
			MPI_Send(&jobs[2*x], 2, MPI_INT, x, NEW_JOB, MPI_COMM_WORLD);
		}
		// dynamically allocate further jobs as results are coming in
		while (resultsCounter < fact)
		{
			//TEST print
			printf("x value %2d, count: %2d\n", x, resultsCounter); fflush(stdout);

			//TODO:
			MPI_Recv(&tempAnswer, 1, MPI_DOUBLE, MPI_ANY_SOURCE, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
			resultsCounter++;
			//TODO:
			answer += tempAnswer;

			// if needed, send next job and increase x
			if (x < fact)
			{
				MPI_Send(&jobs[2*x], 2, MPI_INT, status.MPI_SOURCE, NEW_JOB, MPI_COMM_WORLD);
				x++;
			}
			else
			{
				// notify process about work completion
				MPI_Send(&x, 1, MPI_INT, status.MPI_SOURCE, STOP_WORKING, MPI_COMM_WORLD);  // message with tag==1 from master: work complete
			}
		}
		
		
	} */
	// SLAVES
	/*
	else {  //slaves
		int masterTag = NEW_JOB;
		int jobForBlocks[2];
		while (masterTag == NEW_JOB)
		{
			MPI_Recv(jobForBlocks, 2, MPI_INT, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
			masterTag = status.MPI_TAG;

			if (masterTag == NEW_JOB)
			{
				answer = 0; // make sure local answer == 0
				//TODO:
				//increment_kernel << <blocks, threads, 0, 0 >> >(d_a, value);

				MPI_Send(&answer, 1, MPI_DOUBLE, 0, myid, MPI_COMM_WORLD);	   // report your rank to master in tag (not necessary)
			}
			else
			{
				goto Error;
			}
		}
	}*/

Error:
	//hipFree(d_xya);
	//hipFree(d_kCenters);


	return cudaStatus;
}